#include "hip/hip_runtime.h"


#include <stdio.h>
#include <math.h>

float get_val(int i, int j, int n);
void initialize(float* mat, int size);
void print_mat(float* mat, int size);
bool check_diff(float* mat1, float* mat2, float threshold, int size);
void apply_jacobi(float* mat1, int size);


int main(){

    //////////////////////////////////////
    //Initialization
    //////////////////////////////////////
    int n =512 ;
    float mat[n*n];
    initialize(mat, n);


    //////////////////////////////////////
    //Calculation
    //////////////////////////////////////
    
    apply_jacobi(mat,n);

    //////////////////////////////////////
    //Termination
    //////////////////////////////////////
    return 0;
}

float get_val(int i, int j, int n){
    float common_factor = 4*M_PI/(n - 1 );
    return cos(common_factor * i)*cos(common_factor * j);
}

void initialize(float* mat, int size){

    for(int i=0;i<size;i++){
        for(int j =0; j< size ;j++){
            if(i==0 || i == size-1 || j==0 || j==size-1){
                mat[i*size + j] = 0.0;
            }
            else{
                mat[i*size + j] = get_val(i, j, size);
            }
        }
    }
}


void print_mat(float* mat, int size){
    for(int i=0;i<size;i++){
        for(int j =0; j<size ;j++){
            printf("%f ", mat[i*size + j]);
        }
        printf("\n");
    }
    printf("\n");
}

bool check_diff(float* mat1, float* mat2, float threshold, int size){
    for(int i=1;i<size-1;i++)
        for(int j=1;j<size-1;j++)
            if (fabsf(mat1[i*size + j] - mat2[i*size + j]) > threshold)
                return false;
    
    return true;
}

void apply_jacobi(float* mat, int size){

    float temp[size*size];
    initialize(temp, size);

    while(true){

        for(int i=1; i<size-1;i++){
            for (int j =1; j< size-1;j++){
                float numerator = mat[(i-1)*size + j];
                numerator += mat[(i+1)*size + j];
                numerator += mat[(i)*size + j-1];
                numerator += mat[(i)*size + j+1];
                temp[i*size + j] =  numerator/4;
            }
        }

        if (check_diff(temp, mat, 0.01, size))
            break;

        for(int i=1; i<size-1;i++){
            for (int j =1; j< size-1;j++){
                mat[i*size + j] = temp[i*size + j];
            }
        }

    }
}