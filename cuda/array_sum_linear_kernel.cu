#include "hip/hip_runtime.h"


#include <stdio.h>
#include <math.h>

__global__ void add(int* arr, int size) {
    for(int i=0;i<size;i++) arr[size]+=arr[i];
}
void arraySum(int* arr, int size) {
    int* cuda_arr = nullptr;
    size+=1;
    hipMalloc((void**)&cuda_arr, size * sizeof(int));
    hipMemcpy(cuda_arr, arr, size * sizeof(int), hipMemcpyHostToDevice);
    add<<<1,1>>>(cuda_arr, size-1);
    hipDeviceSynchronize();
    hipMemcpy(arr, cuda_arr, size * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(cuda_arr);
}

int main(int argc, char** argv) {
    const int size = 64;
    int arr[size+1];
    for (int i =0;i< size;i++) arr[i] = 5*i;
    arr[size] = 0;
    // int arr[size+1] = {  1,  2,  3,  4,  5, 6, 7, 8, 0};

    arraySum(arr, size);

    printf("Sum is %d \n", arr[size]);

    hipDeviceReset();

    return 0;
}