#include "hip/hip_runtime.h"


#include <stdio.h>
#include <math.h>

float get_val(int i, int j, int n);
void initialize(float* mat, int size);
void print_mat(float* mat, int size);
// bool check_diff(float* mat1, float* mat2, float threshold, int size);
// void apply_jacobi(float* mat1, int size);

__global__ void cudaInitialize(float* mat, int size){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    // __shared__ float common_factor;
    // common_factor = (4*M_PI/(size - 1 ));
    // printf("%d\n",id);

    if(id >0 && id<size-1){
        for(int j = 0;j<size;j++){
            if(j ==0 || j==size-1)
                mat[id*size + j] =0.0;
            else
                mat[id*size + j] = cos((4*M_PI/(size - 1 )) * (id))*cos((4*M_PI/(size - 1 )) * j);
        }
    }else{
        for(int j = 0;j<size;j++){
            mat[id*size + j] = 0.0;
        }
    }
    __syncthreads();
    for(int j=0;j<size;j++){
        printf("(%d, %f) ",id, mat[id*size + j]);
    }
}

int main(){

    //////////////////////////////////////
    //Initialization
    //////////////////////////////////////
    int n =4 ;
    float mat[n*n];
    // initialize(mat, n);

    //////////////////////////////////////
    //Initializing matrix
    //////////////////////////////////////
    float* cuda_mat = nullptr;
    hipMalloc((void**)&cuda_mat, n * sizeof(float));
    hipMemcpy(cuda_mat, mat, n * sizeof(float), hipMemcpyHostToDevice);
    cudaInitialize<<<1, n>>>(cuda_mat, n);
    hipDeviceSynchronize();
    hipMemcpy(mat, cuda_mat, n * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(cuda_mat);

    print_mat(mat, n);

    //////////////////////////////////////
    //Calculation
    //////////////////////////////////////
    
    // apply_jacobi(mat,n);

    // //////////////////////////////////////
    // //Termination
    // //////////////////////////////////////
    // return 0;
}

float get_val(int i, int j, int n){
    float common_factor = 4*M_PI/(n - 1 );
    return cos(common_factor * i)*cos(common_factor * j);
}

void initialize(float* mat, int size){
    // int id = blockIdx.x * blockDim.x + threadIdx.x;
    for(int i=0;i<size;i++){
        for(int j =0; j< size ;j++){
            if(i==0 || i == size-1 || j==0 || j==size-1){
                mat[i*size + j] = 0.0;
            }
            else{
                mat[i*size + j] = get_val(i, j, size);
            }
        }
    }
}


void print_mat(float* mat, int size){
    for(int i=0;i<size;i++){
        for(int j =0; j<size ;j++){
            printf("%f ", mat[i*size + j]);
        }
        printf("\n");
    }
    printf("\n");
}

// bool check_diff(float* mat1, float* mat2, float threshold, int size){
//     for(int i=1;i<size-1;i++)
//         for(int j=1;j<size-1;j++)
//             if (fabsf(mat1[i*size + j] - mat2[i*size + j]) > threshold)
//                 return false;
    
//     return true;
// }

// __global__ void apply_jacobi(float* mat, int size, float threshold){

//     __shared__ float temp[size*size];
//     __shared__ bool flag;
//     int id = blockIdx.x * blockDim.x + threadIdx.x;

//     for(int j=0;j<size;j++){
//         temp[(id+1)*size + j] = 0.00;
//         temp[j] = 0.00;
//         temp[(size-1)*size + j] = 0.00;
//     }

//     __syncthreads();

//     while(true){

//             flag = true;
//             for (int j =1; j< size-1;j++){
//                 float numerator = mat[(i-1)*size + j];
//                 numerator += mat[(id+1)*size + j];
//                 numerator += mat[(id)*size + j-1];
//                 numerator += mat[(id)*size + j+1];
//                 temp[i*size + j] =  numerator/4;
//             }
        
//         __syncthreads();

//         for(int j = 1;j<size-1;j++){
//             if (mat[id][j] > threshold){
//                 flag = false;
//             } 
//         }

//         __syncthreads();

//         if(!flag) break;


//         // for(int i=1; i<size-1;i++){
//             for (int j =1; j< size-1;j++){
//                 mat[id*size + j] = temp[i*size + j];
//             }
//         // }

//     }
// }