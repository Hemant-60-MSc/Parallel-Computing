#include "hip/hip_runtime.h"


#include <stdio.h>
#include <math.h>


#ifndef __CUDACC__  
    #define __CUDACC__ 
#endif  


__global__ void add(int* arr, int size) {
    int fi = blockIdx.x * blockDim.x + threadIdx.x;
    if (fi< (size/2) ){
        arr[fi] += arr[(size/2) + fi];
    }
    // __sync_threads();
}
void arraySum(int* arr, int size) {
    int* cuda_arr = nullptr;
    hipMalloc((void**)&cuda_arr, size * sizeof(int));

    hipMemcpy(cuda_arr, arr, size * sizeof(int), hipMemcpyHostToDevice);
    int iter_size = size;
    // printf("%d",((int)log2(size)+1));
    for(int i =0;i <= (int)log2(size)+1;i++){
        add<<<2, (iter_size/2 + 1) / 2>>>(cuda_arr, iter_size);
        // cudaDeviceSynchronize();
        // __sync_threads();
        iter_size/=2;
    }

    hipMemcpy(arr, cuda_arr, size * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(cuda_arr);

}

int main(int argc, char** argv) {
    // const int size = 8;
    // int arr[size] = {  1,  2,  3,  4,  5, 6, 7, 8 };

    const int size = 64;
    int arr[size];
    for (int i =0;i< size;i++) arr[i] = 5*i;

    arraySum(arr, size);

    printf("Sum is %d \n", arr[0]);

    hipDeviceReset();

    return 0;
}